#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "HighPassFilter.h"
#include <iostream>

bool isCudaError(hipError_t status)
{
	// printf("[%d] %s\n", status, hipGetErrorString(status));
	return status != hipSuccess;
}

__global__ void kernel(const uint8_t* src, const int loopCnt, uint8_t* dest_1, uint8_t* dest_2, uint8_t* filter_1, uint8_t* filter_2, float* max_1, float* max_2)
{
	const UINT taskIdx = threadIdx.x;
	uint8_t output_1 = 0, output_2 = 0, x1_1 = 0, x1_2 = 0;
	float _max_1 = 0, _max_2 = 0;
	
	for(UINT index = 0; index < loopCnt; index++)
	{
		const UINT realIdx = taskIdx * loopCnt + index;

		dest_1[realIdx] = src[realIdx * 2];
		dest_2[realIdx] = src[realIdx * 2 + 1];

		output_1 = AMPLFAC_1 * (src[realIdx] - x1_1 - output_1 * Y1C_1);
		x1_1 = src[realIdx];
		output_2 = AMPLFAC_2 * (src[realIdx] - x1_2 - output_2 * Y1C_2);
		x1_2 = src[realIdx];

		filter_1[realIdx] = output_1;
		filter_2[realIdx] = output_2;

		if(filter_1[realIdx] > _max_1) _max_1 = filter_1[realIdx];
		if(filter_2[realIdx] > _max_2) _max_2 = filter_2[realIdx];
	}

	max_1[taskIdx] = _max_1;
	max_2[taskIdx] = _max_2;
}

EXPORT int cudaHighPassFilter(const uint8_t* src, const int cnt, uint8_t* dest_1, uint8_t* dest_2, uint8_t* filter_1, uint8_t* filter_2, float* max_1, float* max_2)
{
	printf("in cudaHighPassFilter\n");
	uint8_t *dev_src = 0, *dev_dest_1 = 0, *dev_dest_2 = 0, *dev_filter_1 = 0, *dev_filter_2 = 0;
	float *dev_max_1 = 0, *dev_max_2 = 0;

	hipError_t status;

	// printf("start checkVersion\n");
	// int runtimeVer = 0, driverVer = 0;
	// status = hipRuntimeGetVersion(&runtimeVer);
	// if(isCudaError(status)) goto Exit;
	// status = hipDriverGetVersion(&driverVer);
	// if(isCudaError(status)) goto Exit;

	printf("start cuda\n");
	// printf("cuda runtime ver.%d / cuda driver ver.%d\n", runtimeVer, driverVer);
	status = hipSetDevice(0);
	if(isCudaError(status)) goto Exit;
	printf("success hipSetDevice\n");

	status = hipMalloc((void**)&dev_src, (cnt * 2) * sizeof(uint8_t));
	if (isCudaError(status)) goto Exit;
	status = hipMalloc((void**)&dev_dest_1, cnt * sizeof(uint8_t));
	if (isCudaError(status)) goto Exit;
	status = hipMalloc((void**)&dev_dest_2, cnt * sizeof(uint8_t));
	if (isCudaError(status)) goto Exit;
	status = hipMalloc((void**)&dev_filter_1, cnt * sizeof(uint8_t));
	if (isCudaError(status)) goto Exit;
	status = hipMalloc((void**)&dev_filter_2, cnt * sizeof(uint8_t));
	if (isCudaError(status)) goto Exit;
	status = hipMalloc((void**)&dev_max_1, UNIT_COUNT * sizeof(float));
	if (isCudaError(status)) goto Exit;
	status = hipMalloc((void**)&dev_max_2, UNIT_COUNT * sizeof(float));
	if (isCudaError(status)) goto Exit;
	printf("success hipMalloc\n");

	status = hipMemcpy(dev_src, src, (cnt * 2) * sizeof(uint8_t), hipMemcpyHostToDevice);
	if (isCudaError(status)) goto Exit;
	printf("success hipMemcpy\n");

	kernel<<<1, UNIT_COUNT>>> (dev_src, cnt / UNIT_COUNT, dev_dest_1, dev_dest_2, dev_filter_1, dev_filter_2, dev_max_1, dev_max_2);
	if (isCudaError(hipGetLastError())) goto Exit;
	printf("success kernel\n");

	status = hipDeviceSynchronize();
	if (isCudaError(status)) goto Exit;
	printf("success hipDeviceSynchronize\n");

	status = hipMemcpy(dest_1, dev_dest_1, cnt * sizeof(uint8_t), hipMemcpyDeviceToHost);
	if (isCudaError(status)) goto Exit;
	status = hipMemcpy(dest_2, dev_dest_2, cnt * sizeof(uint8_t), hipMemcpyDeviceToHost);
	if (isCudaError(status)) goto Exit;
	status = hipMemcpy(filter_1, dev_filter_1, cnt * sizeof(uint8_t), hipMemcpyDeviceToHost);
	if (isCudaError(status)) goto Exit;
	status = hipMemcpy(filter_2, dev_filter_2, cnt * sizeof(uint8_t), hipMemcpyDeviceToHost);
	if (isCudaError(status)) goto Exit;
	status = hipMemcpy(max_1, dev_max_1, UNIT_COUNT * sizeof(float), hipMemcpyDeviceToHost);
	if (isCudaError(status)) goto Exit;
	status = hipMemcpy(max_2, dev_max_2, UNIT_COUNT * sizeof(float), hipMemcpyDeviceToHost);
	if (isCudaError(status)) goto Exit;
	printf("success hipMemcpy\n");

Exit:
	hipFree(dev_src);
	hipFree(dev_dest_1);
	hipFree(dev_dest_2);
	hipFree(dev_filter_1);
	hipFree(dev_filter_2);
	hipFree(dev_max_1);
	hipFree(dev_max_2);

	return status;
}